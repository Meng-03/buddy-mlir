#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;
#define CHECK(call)                                                        \
{                                                                          \
    const hipError_t error = call;                                        \
    if (error != hipSuccess)                                              \
    {                                                                      \
        std::cerr << "Error: " << __FILE__ << ", line " << __LINE__       \
                  << "\nCode: " << error << ", reason: " << hipGetErrorString(error) << std::endl; \
        std::exit(error);                                                  \
    }                                                                      \
}
int main(){
  int dev = 0;
  hipDeviceProp_t devProp;
  CHECK(hipGetDeviceProperties(&devProp, dev));
  cout << "使用GPU device " << dev << ": " << devProp.name << std::endl;
  cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
  cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
  cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
  cout << "每个SM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << std::endl;
  cout << "每个SM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
}
// // 输出如下
// 使用GPU device 0: GeForce GT 730
// SM的数量：2
// 每个线程块的共享内存大小：48 KB
// 每个线程块的最大线程数：1024
// 每个SM的最大线程数：2048
// 每个SM的最大线程束数：64
////////////////////
// 使用GPU device 0: Tesla V100-SXM2-32GB
// SM的数量：80
// 每个线程块的共享内存大小：48 KB
// 每个线程块的最大线程数：1024
// 每个SM的最大线程数：2048
// 每个SM的最大线程束数：64