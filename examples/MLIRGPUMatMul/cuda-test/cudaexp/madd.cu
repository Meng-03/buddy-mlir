// cuda中内存管理的API
//  1.device上分配内存的cudaMalloc函数，和释放内存的cudaFree函数
//      cudaError_t cudaMalloc(void **devPtr, size_t size);
//      input:devPtr 指向所分配内存的指针，size一定字节大小的显存
//  2.host和device之间数据通信的cudaMemcpy函数
//      cudaError_t cudaMemcpy(void* dst, const void *src, size_t count, cudaMemcpyKind kind)
//          src指向数据源，
//          dst是目标区域，
//          count是复制的字节数，
//          kind控制复制的方向：cudaMemcpyHostToHost, cudaMemcpyHostToDevice, cudaMemcpyDeviceToHost及cudaMemcpyDeviceToDevice，如cudaMemcpyHostToDevice将host上数据拷贝到device上
// Question:
// - kernel计算的单位是什么呢---是线程thread


#include <hip/hip_runtime.h>
#include <iostream>
//一个线程所执行的
__global__ void add(float* x, float* y,float* z,int n){
    int index=threadIdx.x+blockIdx.x*blockDim.x;
    //步长
    //stride是整个grid的线程数，有时候向量的元素数很多，这时候可以将在每个线程实现多个元素（元素总数/线程总数）的加法，相当于使用了多个grid来处理
    int stride=blockDim.x*gridDim.x;
    for(int i=index;i<n;i+=stride){
        z[i]=x[i]+y[i];
    }
}
int main(){
    int N=1<<20;//2的20次幂
    int nBytes =N*sizeof(float);//n个float所占用的字节数
    
    //申请host内存
    float *a=(float*)malloc(nBytes);
    float *b=(float*)malloc(nBytes);
    float *c=(float*)malloc(nBytes);
    //可以申请托管内存--共同管理host和device中的内存，并自动在host和device中进行数据传输
    // float *a,*b,*c;
    // cudaMallocManaged((void**)&a,nBytes);
    // cudaMallocManaged((void**)&b,nBytes);
    // cudaMallocManaged((void**)&c,nBytes);
    // ...kernel计算完成后
    // //同步device，保证后面的检验能够正确访问
    // //因为kernel和host的执行是异步的，需要用cudaDeviceSynchronize保证device和host同步
    // cudaDeviceSynchronize();


    //初始化数据
    for(int i=0;i<N;i++){
        a[i]=10.0;
        b[i]=20.0;
    }

    //申请device内存
    float *d_a,*d_b,*d_c;
    hipMalloc((void **)&d_a, nBytes);
    hipMalloc((void **)&d_b,nBytes);
    hipMalloc((void **)&d_c,nBytes);

    //将host数据拷贝到device
    hipMemcpy((void*)d_a,(void*)a,nBytes,hipMemcpyHostToDevice);
    hipMemcpy((void*)d_b,(void*)b,nBytes,hipMemcpyHostToDevice);
    //定义kernel的执行配置
    dim3 blockSize(32);
    dim3 gridSize(N+blockSize.x-1/blockSize.x);//+blockSize.x-1为了保证当N不是block的倍数时能够向上取整
    //执行kernel
    add<<<gridSize,blockSize>>>(d_a,d_b,d_c,N);

    //将device结果拷贝到host
    hipMemcpy((void*)c,(void*)d_c,nBytes,hipMemcpyDeviceToHost);

    //检查执行结果
    float maxError=0.0;
    for(int i=0;i<N;i++)
        maxError=fmax(maxError,fabs(c[i]-30.0));
    std::cout<<"最大误差："<<maxError<<std::endl;

    //释放内存
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(a);
    free(b);
    free(c);

    return 0;
}
// nvprof ./madd
//*****block=256
// ==9823== NVPROF is profiling process 9823, command: ./madd
// 最大误差：0
// ==9823== Profiling application: ./madd
// ==9823== Profiling result:
//             Type  Time(%)      Time     Calls       Avg       Min       Max  Name
//  GPU activities:   39.82%  1.7923ms         1  1.7923ms  1.7923ms  1.7923ms  [CUDA memcpy DtoH]
//                    39.37%  1.7718ms         1  1.7718ms  1.7718ms  1.7718ms  add(float*, float*, float*, int)
//                    20.81%  936.38us         2  468.19us  462.85us  473.54us  [CUDA memcpy HtoD]
//       API calls:   95.68%  300.80ms         3  100.27ms  809.34us  299.12ms  cudaMalloc
//                     2.08%  6.5418ms         3  2.1806ms  599.28us  5.1628ms  cudaMemcpy
//                     1.12%  3.5234ms       202  17.442us     210ns  869.81us  cuDeviceGetAttribute
//                     1.01%  3.1834ms         3  1.0611ms  669.79us  1.7633ms  cudaFree
//                     0.09%  282.50us         2  141.25us  127.88us  154.61us  cuDeviceGetName
//                     0.01%  34.645us         1  34.645us  34.645us  34.645us  cudaLaunchKernel
//                     0.00%  10.354us         2  5.1770us  2.3430us  8.0110us  cuDeviceGetPCIBusId
//                     0.00%  2.3430us         3     781ns     297ns  1.6740us  cuDeviceGetCount
//                     0.00%  1.6240us         4     406ns     239ns     737ns  cuDeviceGet
//                     0.00%  1.1340us         2     567ns     484ns     650ns  cuDeviceTotalMem
//                     0.00%     626ns         2     313ns     294ns     332ns  cuDeviceGetUuid