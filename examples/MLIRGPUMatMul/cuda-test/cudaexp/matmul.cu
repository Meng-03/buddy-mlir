//C=A✖️B
//选择grid和block为2-D的--这个结构只是逻辑上的，依据需求去设定

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
int ROW_a=3;
int COL_a=2;
int ROW_b=2;
int COL_b=3;
int ROW_c=3;
int COL_c=3;
__global__ void matmulKernel(float *A, float *B, float *C,int width,int ROW_c,int COL_c){
    int i=threadIdx.x+blockDim.x*blockIdx.x;
    int j=threadIdx.y+blockDim.y*blockIdx.y;
    float ctemp=0.0;
    //2. 这里需要限制越界！！因为block和thread设置的时候会出现越界的情况
    if(i<ROW_c&&j<COL_c){
        // printf("*%d %d\n",i,j);
        for(int k=0;k<width;k++){ 
            ctemp+=A[i*width+k]*B[k*width+j];
        }
        C[i*COL_c+j]=ctemp;//3.这里COL_c写错
    }
}
int main(){
    float *A,*B,*C;
    A=(float*)malloc(sizeof(float)*COL_a*ROW_a);
    B=(float*)malloc(sizeof(float)*COL_b*ROW_b);
    C=(float*)malloc(sizeof(float)*COL_c*ROW_c);

    float *d_A,*d_B,*d_C;
    hipMalloc((void**)&d_A,sizeof(float)*COL_a*ROW_a);
    hipMalloc((void**)&d_B,sizeof(float)*COL_b*ROW_b);
    hipMalloc((void**)&d_C,sizeof(float)*COL_c*ROW_c);

    for(int i=0;i<COL_a*ROW_a;i++){
        A[i]=1.0;
    }
    for(int i=0;i<COL_b*ROW_b;i++){
        B[i]=2.0;
    }

    hipMemcpy((void*)d_A,(void*)A,sizeof(float)*COL_a*ROW_a,hipMemcpyHostToDevice);
    hipMemcpy((void*)d_B,(void*)B,sizeof(float)*COL_b*ROW_b,hipMemcpyHostToDevice);
    
    dim3 blocksize(32,32);
    dim3 gridsize((ROW_c+blocksize.x-1)/blocksize.x,(COL_c+blocksize.y-1)/blocksize.y);
    matmulKernel<<<gridsize,blocksize>>>(d_A,d_B,d_C,COL_a,ROW_c,COL_c);//1.内核调用出错，错传了host端的数据，而不是device端

    hipMemcpy((void*)C,(void*)d_C,sizeof(float)*COL_c*ROW_c,hipMemcpyDeviceToHost);
    
    float maxError=0.0;
    for(int m=0;m<COL_c*ROW_c;m++){
        // int i=m/ROW_c;
        // int j=m%ROW_c;
        // C_ij=0;
        // for(int k=0;k<COL_a;k++)
        //     C_ij+=A[i*width+k]*B[k*width+j];]
        // cout<<C[m]<<endl;
        maxError=fmax(maxError,fabs(C[m]-2*COL_a));
    }
    std::cout<<"最大误差"<<maxError<<std::endl;
    free(A);free(B);free(C);
    hipFree(d_A);hipFree(d_B);hipFree(d_C);
    return 0;

}